#include "hip/hip_runtime.h"
#ifndef MAX_SEARCH_RESULTS
#define MAX_SEARCH_RESULTS 4U
#endif

#define FNV_OFFSET_BASIS 0x811c9dc5

typedef struct {
    uint32_t count;
    struct {
        // One word for gid and 8 for mix hash
        uint32_t gid;
        uint32_t mix[8];
    } result[MAX_SEARCH_RESULTS];
} Search_results;

typedef struct
{
    uint32_t uint32s[32 / sizeof(uint32_t)];
} hash32_t;

// Implementation based on:
// https://github.com/mjosaarinen/tiny_sha3/blob/master/sha3.c


__device__ __constant__ const uint32_t keccakf_rndc[24] = {
    0x00000001, 0x00008082, 0x0000808a, 0x80008000, 0x0000808b, 0x80000001,
    0x80008081, 0x00008009, 0x0000008a, 0x00000088, 0x80008009, 0x8000000a,
    0x8000808b, 0x0000008b, 0x00008089, 0x00008003, 0x00008002, 0x00000080,
    0x0000800a, 0x8000000a, 0x80008081, 0x00008080, 0x80000001, 0x80008008
};

// Implementation of the permutation Keccakf with width 800.
__device__ __forceinline__ void keccak_f800_round(uint32_t st[25], const int r)
{

    const uint32_t keccakf_rotc[24] = {
        1,  3,  6,  10, 15, 21, 28, 36, 45, 55, 2,  14,
        27, 41, 56, 8,  25, 43, 62, 18, 39, 61, 20, 44
    };
    const uint32_t keccakf_piln[24] = {
        10, 7,  11, 17, 18, 3, 5,  16, 8,  21, 24, 4,
        15, 23, 19, 13, 12, 2, 20, 14, 22, 9,  6,  1
    };

    uint32_t t, bc[5];
    // Theta
    for (int i = 0; i < 5; i++)
        bc[i] = st[i] ^ st[i + 5] ^ st[i + 10] ^ st[i + 15] ^ st[i + 20];

    for (int i = 0; i < 5; i++) {
        t = bc[(i + 4) % 5] ^ ROTL32(bc[(i + 1) % 5], 1);
        for (uint32_t j = 0; j < 25; j += 5)
            st[j + i] ^= t;
    }

    // Rho Pi
    t = st[1];
    for (int i = 0; i < 24; i++) {
        uint32_t j = keccakf_piln[i];
        bc[0] = st[j];
        st[j] = ROTL32(t, keccakf_rotc[i]);
        t = bc[0];
    }

    //  Chi
    for (uint32_t j = 0; j < 25; j += 5) {
        for (int i = 0; i < 5; i++)
            bc[i] = st[j + i];
        for (int i = 0; i < 5; i++)
            st[j + i] ^= (~bc[(i + 1) % 5]) & bc[(i + 2) % 5];
    }

    //  Iota
    st[0] ^= keccakf_rndc[r];
}

__device__ __forceinline__ uint32_t cuda_swab32(const uint32_t x)
{
    return __byte_perm(x, x, 0x0123);
}

__device__ __forceinline__ uint64_t seed64_from_seed256(hash32_t seed_256)
{
    return ((uint64_t)cuda_swab32(seed_256.uint32s[0]) << 32) | cuda_swab32(seed_256.uint32s[1]);
}

// Keccak - implemented as a variant of SHAKE
// The width is 800, with a bitrate of 576, a capacity of 224, and no padding
// Only need 64 bits of output for mining
__device__ __noinline__ hash32_t keccak_f800_256(hash32_t header, uint64_t seed, hash32_t digest)
{
    uint32_t st[25];

    for (int i = 0; i < 8; i++)
        st[i] = header.uint32s[i];
    st[8] = seed;
    st[9] = seed >> 32;
    for (int i = 10; i < 18; i++)
        st[i] = digest.uint32s[i - 10];
    for (int i = 18; i < 25; i++)
        st[i] = 0;

    for (int r = 0; r < 22; r++)
        keccak_f800_round(st, r);

    hash32_t temp;

    #pragma unroll
    for (int i = 0; i < 8; i++)
        temp.uint32s[i] = st[i];

    return temp;
}

__device__ __noinline__ uint64_t keccak_f800_64(hash32_t header, uint64_t seed, hash32_t digest)
{
    hash32_t seed_256 = keccak_f800_256(header, seed, digest);
    return seed64_from_seed256(seed_256);
}

#define fnv1a(h, d) (h = (uint32_t(h) ^ uint32_t(d)) * uint32_t(0x1000193))

typedef struct {
    uint32_t z, w, jsr, jcong;
} kiss99_t;

// KISS99 is simple, fast, and passes the TestU01 suite
// https://en.wikipedia.org/wiki/KISS_(algorithm)
// http://www.cse.yorku.ca/~oz/marsaglia-rng.html
__device__ __forceinline__ uint32_t kiss99(kiss99_t &st)
{
    st.z = 36969 * (st.z & 65535) + (st.z >> 16);
    st.w = 18000 * (st.w & 65535) + (st.w >> 16);
    uint32_t MWC = ((st.z << 16) + st.w);
    st.jsr ^= (st.jsr << 17);
    st.jsr ^= (st.jsr >> 13);
    st.jsr ^= (st.jsr << 5);
    st.jcong = 69069 * st.jcong + 1234567;
    return ((MWC^st.jcong) + st.jsr);
}

__device__ __forceinline__ void fill_mix(hash32_t seed, uint32_t lane_id, uint32_t mix[PROGPOW_REGS])
{
    // Use FNV to expand the per-warp seed to per-lane
    // Use KISS to expand the per-lane seed to fill mix
    kiss99_t st;
    uint32_t temp = FNV_OFFSET_BASIS;
    fnv1a(temp, lane_id);
    st.z = fnv1a(temp, seed.uint32s[0 + (lane_id & 1)]);
    st.w = fnv1a(temp, seed.uint32s[2 + (lane_id & 1)]);
    st.jsr = fnv1a(temp, seed.uint32s[4 + (lane_id & 1)]);
    st.jcong = fnv1a(temp, seed.uint32s[6 + (lane_id & 1)]);
    #pragma unroll
    for (int i = 0; i < PROGPOW_REGS; i++)
            mix[i] = kiss99(st);
}

__global__ void 
progpow_search(
    uint64_t start_nonce,
    const hash32_t header,
    const uint64_t target,
    const dag_t *g_dag,
    volatile Search_results* g_output,
    bool hack_false
    )
{
    __shared__ uint32_t c_dag[PROGPOW_CACHE_WORDS];
    uint32_t const gid = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t const nonce = start_nonce + gid;

    const uint32_t lane_id = threadIdx.x & (PROGPOW_LANES - 1);

    // Load the first portion of the DAG into the cache
    for (uint32_t word = threadIdx.x*PROGPOW_DAG_LOADS; word < PROGPOW_CACHE_WORDS; word += blockDim.x*PROGPOW_DAG_LOADS)
    {
        dag_t load = g_dag[word/PROGPOW_DAG_LOADS];
        for(int i=0; i<PROGPOW_DAG_LOADS; i++)
            c_dag[word + i] =  load.s[i];
    }

    hash32_t digest;
    for (int i = 0; i < 8; i++)
        digest.uint32s[i] = 0;
    // keccak(header..nonce)
    hash32_t seed_256 = keccak_f800_256(header, nonce, digest);
    uint64_t seed_64 = seed64_from_seed256(seed_256);

    __syncthreads();

    #pragma unroll 1
    for (uint32_t h = 0; h < PROGPOW_LANES; h++)
    {
        uint32_t mix[PROGPOW_REGS];

        // share the hash's seed across all lanes
        // initialize mix for all lanes
        fill_mix(seed_256, lane_id, mix);

        #pragma unroll 1
        for (uint32_t l = 0; l < PROGPOW_CNT_DAG; l++)
            progPowLoop(l, mix, g_dag, c_dag, hack_false);


        // Reduce mix data to a per-lane 32-bit digest
        uint32_t digest_lane = FNV_OFFSET_BASIS;
        #pragma unroll
        for (int i = 0; i < PROGPOW_REGS; i++)
            fnv1a(digest_lane, mix[i]);

        // Reduce all lanes to a single 256-bit digest
        hash32_t digest_temp;
        #pragma unroll
        for (int i = 0; i < 8; i++)
            digest_temp.uint32s[i] = FNV_OFFSET_BASIS;

        for (int i = 0; i < PROGPOW_LANES; i += 8)
            #pragma unroll
            for (int j = 0; j < 8; j++)
                fnv1a(digest_temp.uint32s[j], SHFL(digest_lane, i + j, PROGPOW_LANES));

        if (h == lane_id)
            digest = digest_temp;
    }

    // keccak(header .. keccak(header..nonce) .. digest);
    if (keccak_f800_64(header, seed_64, digest) > target)
        return;

    uint32_t index = atomicInc((uint32_t *)&g_output->count, 0xffffffff);
    if (index >= MAX_SEARCH_RESULTS)
        return;

    g_output->result[index].gid = gid;
    #pragma unroll
    for (int i = 0; i < 8; i++)
        g_output->result[index].mix[i] = digest.uint32s[i];
}

